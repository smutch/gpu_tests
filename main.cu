#include <mpi.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <sstream>


#define throw_on_cuda_error(cuda_code) \
{ \
_throw_on_cuda_error((cuda_code), __FILE__, __func__, __LINE__); \
}

__host__ void _throw_on_cuda_error(hipError_t cuda_code, const char* file, const char* func, int line)
{
    std::ostringstream mesg;
    mesg << "CUDA error code " << (int)cuda_code << ": " << func << "(" << file << " +" << line << ")";
    if (cuda_code != hipSuccess)
        throw std::runtime_error(mesg.str());
}

void init_cuda()
{
    int world_rank = -1;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    struct hipDeviceProp_t properties;
    int device = -1;

    try {
        // get the number of devices available to this rank
        int num_devices = 0;
        throw_on_cuda_error(hipGetDeviceCount(&num_devices));

        // do your thang to assign this rank to a device
        throw_on_cuda_error(hipSetDevice(world_rank % num_devices));  // alternate assignment between ranks

        // do a check to make sure that we have a working assigned device
        throw_on_cuda_error(hipFree(0));

        // Get the device assigned to this context
        throw_on_cuda_error(hipGetDevice(&device));

        // Get the properties of the device assigned to this context
        throw_on_cuda_error(hipGetDeviceProperties(&properties, device));
    } catch (const std::exception& e) {
        std::cerr << e.what() << std::endl;
    }

    MPI_Barrier(MPI_COMM_WORLD);
    std::cout << "Rank " << world_rank << ": CUDA initialization completed successfully on " << properties.name << "[device=" << device << "] ..." << std::endl;
}

int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
    init_cuda();
    MPI_Finalize();
    return 0;
}
